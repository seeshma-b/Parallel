
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <random>
#include <cstdlib>

#define G 6.674e-11
#define SOFTENING 1e-1

__global__ void compute_forces(int n, double *x, double *y, double *z, double *mass, double *fx, double *fy, double *fz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    double fx_i = 0, fy_i = 0, fz_i = 0;
    for (int j = 0; j < n; ++j) {
        if (i == j) continue;
        double dx = x[j] - x[i];
        double dy = y[j] - y[i];
        double dz = z[j] - z[i];
        double distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        double invDist = rsqrt(distSqr);
        double invDist3 = invDist * invDist * invDist;
        double F = G * mass[i] * mass[j] * invDist3;
        fx_i += dx * F;
        fy_i += dy * F;
        fz_i += dz * F;
    }
    fx[i] = fx_i;
    fy[i] = fy_i;
    fz[i] = fz_i;
}

__global__ void update_positions(int n, double dt, double *x, double *y, double *z, double *vx, double *vy, double *vz, double *fx, double *fy, double *fz, double *mass) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    vx[i] += fx[i] / mass[i] * dt;
    vy[i] += fy[i] / mass[i] * dt;
    vz[i] += fz[i] / mass[i] * dt;
    x[i] += vx[i] * dt;
    y[i] += vy[i] * dt;
    z[i] += vz[i] * dt;
}

void random_init(int n, std::vector<double>& x, std::vector<double>& y, std::vector<double>& z,
                 std::vector<double>& vx, std::vector<double>& vy, std::vector<double>& vz,
                 std::vector<double>& mass) {
    std::mt19937 gen(42);
    std::uniform_real_distribution<double> pos(-1.0, 1.0);
    std::uniform_real_distribution<double> vel(-0.1, 0.1);
    std::uniform_real_distribution<double> m(0.9, 1.1);
    for (int i = 0; i < n; ++i) {
        x[i] = pos(gen); y[i] = pos(gen); z[i] = pos(gen);
        vx[i] = vel(gen); vy[i] = vel(gen); vz[i] = vel(gen);
        mass[i] = m(gen);
    }
}

int main(int argc, char* argv[]) {
    if (argc != 6) {
        std::cerr << "Usage: " << argv[0] << " <num_particles> <dt> <steps> <print_interval> <block_size>\n";
        return 1;
    }

    int n = atoi(argv[1]);
    double dt = atof(argv[2]);
    int steps = atoi(argv[3]);
    int print_interval = atoi(argv[4]);
    int blockSize = atoi(argv[5]);

    std::vector<double> x(n), y(n), z(n);
    std::vector<double> vx(n), vy(n), vz(n);
    std::vector<double> fx(n), fy(n), fz(n);
    std::vector<double> mass(n);

    random_init(n, x, y, z, vx, vy, vz, mass);

    double *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_fx, *d_fy, *d_fz, *d_mass;
    size_t bytes = n * sizeof(double);

    hipMalloc(&d_x, bytes); hipMalloc(&d_y, bytes); hipMalloc(&d_z, bytes);
    hipMalloc(&d_vx, bytes); hipMalloc(&d_vy, bytes); hipMalloc(&d_vz, bytes);
    hipMalloc(&d_fx, bytes); hipMalloc(&d_fy, bytes); hipMalloc(&d_fz, bytes);
    hipMalloc(&d_mass, bytes);

    hipMemcpy(d_x, x.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_z, z.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vx, vx.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vy, vy.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vz, vz.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_mass, mass.data(), bytes, hipMemcpyHostToDevice);

    int gridSize = (n + blockSize - 1) / blockSize;

    for (int step = 0; step < steps; ++step) {
        compute_forces<<<gridSize, blockSize>>>(n, d_x, d_y, d_z, d_mass, d_fx, d_fy, d_fz);
        update_positions<<<gridSize, blockSize>>>(n, dt, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_fx, d_fy, d_fz, d_mass);
        if (step % print_interval == 0) {
            hipMemcpy(x.data(), d_x, bytes, hipMemcpyDeviceToHost);
            std::cout << n << '\t';
            for (int i = 0; i < n; ++i) std::cout << x[i] << '\t';
            std::cout << '\n';
        }
    }

    hipFree(d_x); hipFree(d_y); hipFree(d_z);
    hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
    hipFree(d_fx); hipFree(d_fy); hipFree(d_fz);
    hipFree(d_mass);

    return 0;
}
